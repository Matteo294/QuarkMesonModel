#include "hip/hip_runtime.h"
#include <array>
#include <complex>
#include <iostream>
#include <thrust/complex.h>
#include <fstream>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include "../Dirac.cuh"
#include "../Spinor.cuh"
#include "../Lattice.cuh"
#include "../params.h"

namespace cg = cooperative_groups;

using std::conj;

double const q = 6.0 * (double) M_PI / Nx;
double const p = 7.0 * (double) M_PI / Nt;

thrust::complex<double> im {0.0, 1.0};

using cpdouble = thrust::complex<double>;

template <typename T>
__host__ void CGsolver_solve_D(Spinor<T> *inVec, Spinor<T> *outVec, DiracOP<T>& D, thrust::complex<double> *M1, thrust::complex<double> *M2, thrust::complex<double> *M3, thrust::complex<double> *M4, int const numBlocks, int const numThreads);

__global__ void gpuDotProduct(cpdouble *vecA, cpdouble *vecB, cpdouble *result, int size);

		
int main() {

	thrust::complex<double> *M1, *M2, *M3, *M4;
	Spinor<double> *in, *out;
	Lattice lattice(Nt, Nx);
	DiracOP<double> Dirac(fermion_mass, g_coupling, lattice);

	// Allocate two vectors and mesons matrix
	hipMallocManaged(&M1, sizeof(thrust::complex<double>) * lattice.vol);
	hipMallocManaged(&M2, sizeof(thrust::complex<double>) * lattice.vol);
	hipMallocManaged(&M3, sizeof(thrust::complex<double>) * lattice.vol);
	hipMallocManaged(&M4, sizeof(thrust::complex<double>) * lattice.vol);
	hipMallocManaged(&in, sizeof(Spinor<double>) * lattice.vol);
	hipMallocManaged(&out, sizeof(Spinor<double>) * lattice.vol);
	
	for(int i=0; i<lattice.vol; i++){
		M1[i] = sigma + im * pi[2];
		M4[i] = sigma - im * pi[2];
		M2[i] = im * (pi[0] - im * pi[1]);
		M3[i] = im * (pi[0] + im * pi[1]);
	}

	for(int i=0; i<lattice.vol; i++){in[i].setZero(); out[i].setZero();}

	// set source
	for(int i=0; i<lattice.vol; i++){
		auto idx = lattice.eoToVec(i);
		in[i].val[0] = 1.0 * exp(im*idx[1]*q+im*idx[0]*p);
		//in[i].val[1] = 1.0 * exp(im*idx[1]*q+im*idx[0]*p);
	}
	thrust::complex<double> xtest = in[1].val[0];

	MatrixType useDagger = MatrixType::Normal;
	// diagArgs should be passed to all the diagonal (in spacetime) functions: Doo, Dee, Dooinv, Deeinv
	void *diagArgs[] = {(void*)&in, (void*)&out, (void*) &lattice.vol, (void*) &fermion_mass, (void*) &g_coupling, (void*)&useDagger, (void*)&M1, (void*)&M2, (void*)&M3, (void*)&M4};
	// hopping should be passed to all the off-diagonal (in spacetime) functions: Deo, Doe
	void *hoppingArgs[] = {(void*)&in, (void*) &out, (void*) &lattice.vol, (void*) &useDagger, (void*) &lattice.IUP, (void*) &lattice.IDN};


	int numBlocks = 0;
	int numThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&numBlocks, &numThreads, gpuDotProduct);
	hipDeviceSynchronize();

	useDagger = MatrixType::Normal;
	diagArgs[0] = (void*) &in; diagArgs[1] = (void*) &out;
	hoppingArgs[0] = (void*) &in; hoppingArgs[1] = (void*) &out;
	for(int i=0; i<lattice.vol; i++){out[i].setZero();}
	Dirac.applyD(diagArgs, hoppingArgs);
	hipDeviceSynchronize();

	std::ofstream myfile;
	myfile.open("planewave.csv");
	myfile << "nt,nx,v1,v2,v3,v4" << std::endl;
	int i;
	thrust::complex<double> v, w;
	cpdouble r = 0.0;
	for(int nx=0; nx<Nx; nx++){
		for(int nt=0; nt<Nt; nt++){
			i = lattice.toEOflat(nt, nx);
			v = (fermion_mass + g_coupling*sigma + 2.0*pow(sin(0.5*p), 2) + im * sin(p) + 2.0 * pow(sin(0.5*q), 2)) * in[i].val[0] + (-g_coupling*pi[2] + im*sin(q)) * in[i].val[1];
			w = (fermion_mass + g_coupling*sigma + 2.0*pow(sin(0.5*p), 2) - im * sin(p) + 2.0 * pow(sin(0.5*q), 2)) * in[i].val[1] + ( g_coupling*pi[2] + im*sin(q)) * in[i].val[0];
			/*myfile  	<< nt << "," << nx << "," << out[i].val[0].real() << "," << out[i].val[1].real() << ","
						<< v.real() << ","
						<< w.real() << "\n";*/
			std::cout 	<< "Site nt=" << nt << " nx=" << nx << "\n"  
						<< "1st component \t --> \t measured: " << out[i].val[0] << "\t\t expected :" << v << "\n"
						<< "2nd component \t --> \t measured: " << out[i].val[1] << "\t\t expected :" << w << "\n\n";
		}
	}


	std::cout << out[1].val[0] << std::endl;
	std::cout << xtest << std::endl;
 
	std::cout << "Last error: " << hipGetLastError() << ": " << hipGetErrorString(hipGetLastError()) << "\n";

	hipFree(M1);
	hipFree(M2);
	hipFree(M3);
	hipFree(M4);
	hipFree(in);
	hipFree(out);
	
	return 0;
}


__global__ void gpuDotProduct(cpdouble *vecA, cpdouble *vecB, cpdouble *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	*result = 0.0;
	extern __shared__ cpdouble tmp[];

	cpdouble temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<cpdouble>());

	if (tile32.thread_rank() == 0) {
		tmp[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<cpdouble>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}


template <typename T>
__host__ void CGsolver_solve_D(Spinor<T> *inVec, Spinor<T> *outVec, DiracOP<T>& D, thrust::complex<double> *M1, thrust::complex<double> *M2, thrust::complex<double> *M3, thrust::complex<double> *M4, int const numBlocks, int const numThreads){	
	
	int const vol = D.lattice.vol;
	int mySize = D.lattice.vol * 4;

	Spinor<T> *r, *p, *temp, *temp2; // allocate space ?? 
	thrust::complex<T> alpha; // allocate space ??
	T beta, rmodsq;
	cpdouble *dot_res;

	hipMallocManaged(&r, sizeof(Spinor<T>) * vol);
	hipMallocManaged(&p, sizeof(Spinor<T>) * vol);
	hipMallocManaged(&temp, sizeof(Spinor<T>) * vol);
	hipMallocManaged(&temp2, sizeof(Spinor<T>) * vol);
	hipMallocManaged(&dot_res, sizeof(cpdouble));

	for(int i=0; i<vol; i++) {
		outVec[i] = Spinor<T> ();
		temp[i] = Spinor<T> ();
		temp2[i] = Spinor<T> ();
		for(int j=0; j<4; j++) r[i].val[j] = inVec[i].val[j];
		for(int j=0; j<4; j++) p[i].val[j] = r[i].val[j];
	}

	// Set up dot product call
	void *dotArgs[] = {(void*) &r, (void*) &r, (void*) &dot_res, (void*) &mySize};
	auto dimGrid = dim3(numBlocks, 1, 1);
	auto dimBlock = dim3(numThreads, 1, 1);

	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid, dimBlock, dotArgs, sizeof(cpdouble) * ((mySize/32/numBlocks) + 1), NULL);
	hipDeviceSynchronize();
	rmodsq = dot_res->real();

	MatrixType dag = MatrixType::Normal;

	void *diagArgs[] = {(void*)&p, (void*)&temp2, (void*) &D.lattice.vol, (void*) &fermion_mass, (void*) &g_coupling, (void*)&dag, (void*)&M1, (void*)&M2, (void*)&M3, (void*)&M4};
	void *hoppingArgs[] = {(void*)&p, (void*)&temp2, (void*) &D.lattice.vol, (void*)&dag, (void*)&D.lattice.IUP, (void*)&D.lattice.IDN};

	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) temp[i].val[j] = 2.0 * p[i].val[j];
		}

		// Set buffers to zero to store the result fo the Dirac operator applied to p
		for(int i=0; i<D.lattice.vol; i++) {temp2[i].setZero(); temp[i].setZero();}

		// Apply D dagger
		dag = MatrixType::Dagger;
		diagArgs[0] = (void*) &p; diagArgs[1] = (void*) &temp2;
		hoppingArgs[0] = (void*) &p; hoppingArgs[1] = (void*) &temp2;
		D.applyD(diagArgs, hoppingArgs);
		// Apply D
		dag = MatrixType::Normal;
		diagArgs[0] = (void*) &temp2; diagArgs[1] = (void*) &temp;
		hoppingArgs[0] = (void*) &temp2; hoppingArgs[1] = (void*) &temp;
		D.applyD(diagArgs, hoppingArgs);
		
		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid, dimBlock, dotArgs, sizeof(cpdouble) * ((mySize/32/numBlocks) + 1), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 

		// x = x + alpha p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) outVec[i].val[j] += alpha*p[i].val[j];
		}
		// r = r - alpha A p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) r[i].val[j] -= alpha*temp[i].val[j];
		}

		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid, dimBlock, dotArgs, sizeof(cpdouble) * ((mySize/32/numBlocks) + 1), NULL);
		hipDeviceSynchronize();
		beta = dot_res->real() / rmodsq;

		// p = r - beta p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) p[i].val[j] = r[i].val[j] + beta*p[i].val[j];
		}

		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid, dimBlock, dotArgs, sizeof(cpdouble) * ((mySize/32/numBlocks) + 1), NULL);
		hipDeviceSynchronize();
		rmodsq = dot_res->real();
	}

	if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";

	hipFree(r);
	hipFree(p);
	hipFree(temp);
	hipFree(temp2);
	hipFree(dot_res);
}