#include "hip/hip_runtime.h"
#include "CGsolver.cuh"


__device__ void applyD(cp<double> *in, cp<double> *out, int vol){
    auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
		out[i] = ((cp<double>) i + 0.1) * in[i];
	}
}


CGsolver::CGsolver(){
	hipMallocManaged(&dot_res, sizeof(thrust::complex<double>));
	hipMallocManaged(&rmodsq, sizeof(double));
	hipMallocManaged(&alpha, sizeof(thrust::complex<double>));
	hipMallocManaged(&beta, sizeof(thrust::complex<double>));
        
	// Arguments are assigned in the following, but then the pointers are modified before each kernel call
	dotArgs[0] = (void*) &(r.data()); 
	dotArgs[1] = (void*) &(r.data()); 
	dotArgs[2] = (void*) &dot_res; 
	dotArgs[3] = (void*) &myvol;
	
	setZeroArgs[0] = (void*) &temp.data(); 
	setZeroArgs[1] = (void*) &myvol;
	
	sumArgs[0] = (void*) &r.data(); 
	sumArgs[1] = (void*) &r.data(); 
	sumArgs[2] = (void*) &r.data(); 
	sumArgs[3] = (void*) &beta;
	sumArgs[4] = (void*) &myvol;
    
	copyArgs[0] = (void*) &r.data(); 
	copyArgs[1] = (void*) &r.data(); 
	copyArgs[2] = (void*) &myvol;

	myvol = 4*vol;

	
}

void CGsolver::solve(cp<double>  *inVec, cp<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
    auto MatType = Mtype;
    int nBlocks = 0;
	int nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, solve_kernel);
	hipDeviceSynchronize();
	auto dimGrid = dim3(nBlocks, 1, 1);
	auto dimBlock = dim3(nThreads, 1, 1);
    int sMemSize = sizeof(thrust::complex<double>) * (nThreads/32);
    void *solveArgs[] = {(void*) &inVec, (void*) &outVec, 
						(void*) &temp.data(), (void*) &temp2.data(), (void*) &r.data(), (void*) &p.data(), 
						(void*) &alpha, (void*)&beta,
						(void*) &D.M, 
						(void*) &D.EO2N, (void*) &D.IUP, (void*)&D.IDN,
						(void*) &MatType, (void*)&dot_res, (void*)&rmodsq};
    hipLaunchCooperativeKernel((void*) solve_kernel, dimGrid, dimBlock, solveArgs, sMemSize, NULL);
	hipDeviceSynchronize();
}

__device__ void setZeroGPU(thrust::complex<double> *v, int const vol){
	cg::grid_group grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()) v[i] = 0.0;
}

__device__ void copyVec(thrust::complex<double> *v1,thrust::complex<double> *v2, int const vol){
	cg::grid_group grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()) v1[i] = v2[i];
}


__global__ void solve_kernel(cp<double>  *inVec, cp<double> *outVec, 
                             cp<double> *temp, cp<double> *temp2, cp<double> *r, cp<double> *p,
							 cp<double> *alpha, cp<double> *beta,
							 double *M, 
							 int *EO2N, my2dArray *IUP, my2dArray *IDN,  
							 MatrixType Mtype, cp<double> *dot_res, double *rmodsq)
{
        
    auto grid = cg::this_grid();
	int myvol = 4*vol;
    
    setZeroGPU(outVec, myvol);
    cg::sync(grid);
    setZeroGPU(temp, myvol);
    cg::sync(grid);
    setZeroGPU(temp2, myvol);
    cg::sync(grid);
    
    copyVec(r, inVec, myvol);
    cg::sync(grid);
    copyVec(p, inVec, myvol);
    cg::sync(grid);
    

	if (threadIdx.x == 0 && blockIdx.x == 0) *dot_res = 0.0;
    gpuDotProduct(r, r, dot_res, myvol);
    cg::sync(grid);

    *rmodsq = abs(*dot_res);

	auto MatType = Mtype;

	cg::sync(grid); 

    int k;
	for(k=0; k<IterMax && sqrt(*rmodsq) > tolerance; k++){

		setZeroGPU(temp, myvol);
        cg::sync(grid);
        setZeroGPU(temp2, myvol);

  		// Apply D dagger
		if (Mtype == MatrixType::Normal) MatType = MatrixType::Dagger;
		else MatType = MatrixType::Normal;
        cg::sync(grid);
        D_oo(p, temp2, MatType, M, EO2N);
        cg::sync(grid);
        D_ee(p, temp2, MatType, M, EO2N);
        cg::sync(grid);
        D_eo(p, temp2, MatType, IUP, IDN);
        cg::sync(grid);
        D_oe(p, temp2, MatType, IUP, IDN);

		// Apply D
		if (Mtype == MatrixType::Normal) MatType = MatrixType::Normal;
		else MatType = MatrixType::Dagger;
        cg::sync(grid);
        D_oo(temp2, temp, MatType, M, EO2N);
        cg::sync(grid);
        D_ee(temp2, temp, MatType, M, EO2N);
        cg::sync(grid);
        D_eo(temp2, temp, MatType, IUP, IDN);
        cg::sync(grid);
        D_oe(temp2, temp, MatType, IUP, IDN);

    
		if (threadIdx.x == 0 && blockIdx.x == 0) *dot_res = 0.0;
        cg::sync(grid);
        gpuDotProduct(p, temp, dot_res, myvol);
        cg::sync(grid);
        
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			*alpha = *rmodsq / *dot_res; 
		}


		// x = x + alpha p
		cg::sync(grid);
        gpuSumSpinors(outVec, p, outVec, *alpha, myvol);
		
		// r = r - alpha A p
		if (threadIdx.x == 0 && blockIdx.x == 0) *alpha = - *alpha;
		cg::sync(grid);
        gpuSumSpinors(r, temp, r, *alpha, myvol);

        if (threadIdx.x == 0 && blockIdx.x == 0) *dot_res = 0.0;
		cg::sync(grid);
        gpuDotProduct(r, r, dot_res, myvol);
        cg::sync(grid);
        
		if (threadIdx.x == 0 && blockIdx.x == 0){
			*beta = abs(*dot_res) / *rmodsq;
        	*rmodsq = abs(*dot_res);
		}

		// p = r - beta p
		cg::sync(grid);
        gpuSumSpinors(r, p, p, *beta, myvol);
        cg::sync(grid);

	
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
	//if (k >= IterMax) std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
}

__device__ void gpuSumSpinors(cp<double> *s1, cp<double> *s2, cp<double> *res, thrust::complex<double> c, int size){
	auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < size; i += grid.size()){
		res[i] = s1[i] + c * s2[i];
	}
}





/*void CGsolver::solveEO(cp<double>  *inVec, cp<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
	double rmodsq;

	myvol = 2*vol;
	setZeroArgs[0] = (void*) &(temp.data());
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &temp2.data();
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &outVec;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &r.data(); copyArgs[1] = (void*) &inVec;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &p.data(); copyArgs[1] = (void*) &r.data();
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();

	*dot_res = 0.0;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = abs(*dot_res);


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		setZeroArgs[0] = (void*) &temp.data();
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();
		setZeroArgs[0] = (void*) &temp2.data();
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();

		// Apply D dagger
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Dagger);
		else D.setDagger(MatrixType::Normal);
		D.setInVec(p.data());
		D.setOutVec(temp2.data());
		D.applyDhat();
		// Apply D
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Normal);
		else D.setDagger(MatrixType::Dagger);
		D.setInVec(temp2.data());
		D.setOutVec(temp.data());
		D.applyDhat();
		
		dotArgs[0] = (void*) &p.data(); dotArgs[1] = (void*) &temp.data();

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 


		// x = x + alpha p
		sumArgs[0] = (void*) &outVec;
		sumArgs[1] = (void*) &p.data();
		sumArgs[2] = (void*) &outVec;
		sumArgs[3] = (void*) &alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		
		// r = r - alpha A p
		sumArgs[0] = (void*) &r.data();
		sumArgs[1] = (void*) &temp.data();
		sumArgs[2] = (void*) &r.data();
		sumArgs[3] = (void*) &alpha;
		alpha = -alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();


		dotArgs[0] = (void*) &r.data(); dotArgs[1] = (void*) &r.data();
		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;
        rmodsq = abs(*dot_res);

		// p = r - beta p
		sumArgs[0] = (void*) &r.data();
		sumArgs[1] = (void*) &p.data();
		sumArgs[2] = (void*) &p.data();
		sumArgs[3] = (void*) &beta;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
	
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
	if (k >= IterMax) std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
}*/

/*__global__ void gpuDotProduct(thrust::complex<double> *vecA, thrust::complex<double> *vecB, thrust::complex<double> *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	extern __shared__ thrust::complex<double> tmp2[];

	thrust::complex<double> temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

	if (tile32.thread_rank() == 0) {
		tmp2[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp2[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}


__global__ void gpuSumSpinors(cp<double> *s1, cp<double> *s2, cp<double> *res, thrust::complex<double> c, int size){
	auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < size; i += grid.size()){
		res[i] = s1[i] + c * s2[i];
	}
}*/
