#include "hip/hip_runtime.h"
#include "CGsolver.cuh"


CGsolver::CGsolver(){
    hipMallocManaged(&r, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&p, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp2, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&sol, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&dot_res, sizeof(thrust::complex<double>));

    int numBlocks = 0;
	int numThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&numBlocks, &numThreads, gpuDotProduct);
	hipDeviceSynchronize();
	dimGrid_dot = dim3(numBlocks, 1, 1);
	dimBlock_dot = dim3(numThreads, 1, 1);

	/*numBlocks = 0;
	numThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&numBlocks, &numThreads, setSpinorToZero);
	hipDeviceSynchronize();
	dimGrid_setZero = dim3(numBlocks, 1, 1);
	dimBlock_setZero = dim3(numThreads, 1, 1);*/

	
}

CGsolver::~CGsolver(){
	hipFree(r);
	hipFree(p);
	hipFree(temp);
	hipFree(temp2);
	hipFree(dot_res);
	hipFree(sol);
}



void CGsolver::solve(Spinor<double>  *inVec, Spinor<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
	//hipMemcpy(inVec, r, vol * sizeof(Spinor<double>), hipMemcpyHostToDevice);
	double rmodsq;

	void *dotArgs[] = {(void*) &r, (void*) &r, (void*) &dot_res, (void*) &spinor_vol};
	// set up set spinor to zero
	int nBlocks = 0;
	int nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, setZeroGPU);
	hipDeviceSynchronize();
	nBlocks = 0;
	nThreads = 0;
	dimGrid_zero = dim3(nBlocks, 1, 1);
	dimBlock_zero = dim3(nThreads, 1, 1);
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, gpuSumSpinors);
	hipDeviceSynchronize();
	dimGrid_sum = dim3(nBlocks, 1, 1);
	dimBlock_sum = dim3(nThreads, 1, 1);
	void *setZeroArgs[] = {(void*)temp, (void*) &spinor_vol};
	void *sumArgs[] = {(void*) &r, (void*) &r, (void*) &r, (void*) &beta};

	for(int i=0; i<vol; i++) {
		outVec[i] = Spinor<double> ();
		temp[i] = Spinor<double> ();
		temp2[i] = Spinor<double> ();
		for(int j=0; j<4; j++){
			r[i].val[j] = inVec[i].val[j];
			p[i].val[j] = r[i].val[j];
		}
	}

	/*setZeroArgs[0] = (void*) &temp;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &temp2;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &sol;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();*/


	*dot_res = 0.0;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = abs(*dot_res);
	std::cout << "Initial rmodsq: " << rmodsq << std::endl;


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		// Set buffers to zero to store the result fo the Dirac operator applied to p
		setZeroArgs[0] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();
		setZeroArgs[0] = (void*) &temp2;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();

		// Apply D dagger
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Dagger);
		else D.setDagger(MatrixType::Normal);
		D.setInVec(p);
		D.setOutVec(temp2);
		D.applyD();
		// Apply D
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Normal);
		else D.setDagger(MatrixType::Dagger);
		D.setInVec(temp2);
		D.setOutVec(temp);
		D.applyD();
		
		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 

		// x = x + alpha p
		sumArgs[0] = (void*) &outVec;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &outVec;
		sumArgs[3] = (void*) &alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		/*for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) outVec[i].val[j] += alpha*p[i].val[j];
		}*/
		// r = r - alpha A p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &temp;
		sumArgs[2] = (void*) &r;
		sumArgs[3] = (void*) &alpha;
		alpha = -alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		/*for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) r[i].val[j] -= alpha*temp[i].val[j];
		}*/

		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;

		// p = r - beta p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &p;
		sumArgs[3] = (void*) &beta;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		/*for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) p[i].val[j] = r[i].val[j] + beta*p[i].val[j];
		}*/

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		rmodsq = abs(*dot_res);
	}

	if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";

}

__global__ void gpuDotProduct(thrust::complex<double> *vecA, thrust::complex<double> *vecB, thrust::complex<double> *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	//*result = 0.0;
	extern __shared__ thrust::complex<double> tmp2[];

	thrust::complex<double> temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

	if (tile32.thread_rank() == 0) {
		tmp2[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp2[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}


__global__ void gpuSumSpinors(Spinor<double> *s1, Spinor<double> *s2, Spinor<double> *res, thrust::complex<double> c){
	auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
		for(int j=0; j<4; j++) res[i].val[j] = s1[i].val[j] + c * s2[i].val[j];
	}
}
