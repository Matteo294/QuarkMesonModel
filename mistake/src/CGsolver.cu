#include "hip/hip_runtime.h"
#include "CGsolver.cuh"


CGsolver::CGsolver(){
    hipMallocManaged(&r, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&p, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp2, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&dot_res, sizeof(thrust::complex<double>));

    int nBlocks = 0;
	int nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, gpuDotProduct);
	hipDeviceSynchronize();
	dimGrid_dot = dim3(nBlocks, 1, 1);
	dimBlock_dot = dim3(nThreads, 1, 1);
    
    nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, setZeroGPU);
	hipDeviceSynchronize();
    dimGrid_zero = dim3(nBlocks, 1, 1);
	dimBlock_zero = dim3(nThreads, 1, 1);
    
	nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, gpuSumSpinors);
	hipDeviceSynchronize();
	dimGrid_sum = dim3(nBlocks, 1, 1);
	dimBlock_sum = dim3(nThreads, 1, 1);
    
    nBlocks = 0;
	nThreads = 0;
	dimGrid_copy = dim3(nBlocks, 1, 1);
	dimBlock_copy = dim3(nThreads, 1, 1);
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, copyVec);
	hipDeviceSynchronize();
	dimGrid_copy = dim3(nBlocks, 1, 1);
	dimBlock_copy = dim3(nThreads, 1, 1);

	
}

CGsolver::~CGsolver(){
	hipFree(r);
	hipFree(p);
	hipFree(temp);
	hipFree(temp2);
	hipFree(dot_res);
}



void CGsolver::solve(Spinor<double>  *inVec, Spinor<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
	double rmodsq;
    
    void *dotArgs[] = {(void*) &r, (void*) &r, (void*) &dot_res, (void*) &spinor_vol};
	void *setZeroArgs[] = {(void*)temp, (void*) &spinor_vol};
	void *sumArgs[] = {(void*) &r, (void*) &r, (void*) &r, (void*) &beta};
    void *copyArgs[] = {(void*) &r, (void*) &inVec, (void*) &spinor_vol};

	setZeroArgs[0] = (void*) &temp;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &temp2;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &outVec;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &r; copyArgs[1] = (void*) &inVec;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &p; copyArgs[1] = (void*) &r;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();


	*dot_res = 0.0;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = abs(*dot_res);


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		setZeroArgs[0] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();
		setZeroArgs[0] = (void*) &temp2;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();

		// Apply D dagger
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Dagger);
		else D.setDagger(MatrixType::Normal);
		D.setInVec(p);
		D.setOutVec(temp2);
		D.applyD();
		// Apply D
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Normal);
		else D.setDagger(MatrixType::Dagger);
		D.setInVec(temp2);
		D.setOutVec(temp);
		D.applyD();
		
		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 


		// x = x + alpha p
		sumArgs[0] = (void*) &outVec;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &outVec;
		sumArgs[3] = (void*) &alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		
		// r = r - alpha A p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &temp;
		sumArgs[2] = (void*) &r;
		sumArgs[3] = (void*) &alpha;
		alpha = -alpha;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();


		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;
        rmodsq = abs(*dot_res);

		// p = r - beta p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &p;
		sumArgs[3] = (void*) &beta;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
	
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";

}

__global__ void gpuDotProduct(thrust::complex<double> *vecA, thrust::complex<double> *vecB, thrust::complex<double> *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	//*result = 0.0;
	extern __shared__ thrust::complex<double> tmp2[];

	thrust::complex<double> temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

	if (tile32.thread_rank() == 0) {
		tmp2[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp2[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}


__global__ void gpuSumSpinors(Spinor<double> *s1, Spinor<double> *s2, Spinor<double> *res, thrust::complex<double> c){
	auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
		for(int j=0; j<4; j++) res[i].val[j] = s1[i].val[j] + c * s2[i].val[j];
	}
}
