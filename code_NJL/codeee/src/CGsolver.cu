#include "hip/hip_runtime.h"
#include "CGsolver.cuh"


CGsolver::CGsolver(){
    hipMallocManaged(&r, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&p, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp2, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&dot_res, sizeof(thrust::complex<double>));

    int nBlocks = 0;
	int nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, gpuDotProduct);
	hipDeviceSynchronize();
	dimGrid_dot = dim3(nBlocks, 1, 1);
	dimBlock_dot = dim3(nThreads, 1, 1);
    
    nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, setZeroGPU);
	hipDeviceSynchronize();
    dimGrid_zero = dim3(nBlocks, 1, 1);
	dimBlock_zero = dim3(nThreads, 1, 1);
    
	nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, gpuSumSpinors);
	hipDeviceSynchronize();
	dimGrid_sum = dim3(nBlocks, 1, 1);
	dimBlock_sum = dim3(nThreads, 1, 1);
    
    nBlocks = 0;
	nThreads = 0;
	dimGrid_copy = dim3(nBlocks, 1, 1);
	dimBlock_copy = dim3(nThreads, 1, 1);
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, copyVec);
	hipDeviceSynchronize();
	dimGrid_copy = dim3(nBlocks, 1, 1);
	dimBlock_copy = dim3(nThreads, 1, 1);

	// Arguments are assigned in the following, but then the pointers are modified before each kernel call
	dotArgs[0] = (void*) &r; 
	dotArgs[1] = (void*) &r; 
	dotArgs[2] = (void*) &dot_res; 
	dotArgs[3] = (void*) &myvol;
	
	setZeroArgs[0] = (void*) &temp; 
	setZeroArgs[1] = (void*) &myvol;
	
	sumArgs[0] = (void*) &r; 
	sumArgs[1] = (void*) &r; 
	sumArgs[2] = (void*) &r; 
	sumArgs[3] = (void*) &beta;
	sumArgs[4] = (void*) &myvol;
    
	copyArgs[0] = (void*) &r; 
	copyArgs[1] = (void*) &r; 
	copyArgs[2] = (void*) &myvol;

	myvol = 4*vol;

	
}

CGsolver::~CGsolver(){
	hipFree(r);
	hipFree(p);
	hipFree(temp);
	hipFree(temp2);
	hipFree(dot_res);
}



void CGsolver::solve(Spinor<double>  *inVec, Spinor<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
	double rmodsq;

	myvol = 4*vol;
	setZeroArgs[0] = (void*) &temp;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &temp2;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &outVec;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &r; copyArgs[1] = (void*) &inVec;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &p; copyArgs[1] = (void*) &r;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();

	*dot_res = 0.0;
	myvol = 4*vol;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = abs(*dot_res);


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		myvol = 4*vol;
		setZeroArgs[0] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();
		setZeroArgs[0] = (void*) &temp2;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();

		// Apply D dagger
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Dagger);
		else D.setDagger(MatrixType::Normal);
		D.setInVec(p);
		D.setOutVec(temp2);
		D.applyD();
		// Apply D
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Normal);
		else D.setDagger(MatrixType::Dagger);
		D.setInVec(temp2);
		D.setOutVec(temp);
		D.applyD();

		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;

		*dot_res = 0.0;
		myvol = 4*vol;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 

		// x = x + alpha p
		sumArgs[0] = (void*) &outVec;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &outVec;
		sumArgs[3] = (void*) &alpha;
		myvol = vol;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		
		// r = r - alpha A p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &temp;
		sumArgs[2] = (void*) &r;
		sumArgs[3] = (void*) &alpha;
		alpha = -alpha;
		myvol = vol;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();


		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		*dot_res = 0.0;
		myvol = 4*vol;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;
        rmodsq = abs(*dot_res);

		// p = r - beta p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &p;
		sumArgs[3] = (void*) &beta;
		myvol = vol;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
	
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
	if (k >= IterMax) std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
}

/*void CGsolver::solveEO(Spinor<double>  *inVec, Spinor<double> *outVec, DiracOP<double>& D, MatrixType Mtype){
	double rmodsq;

	int const vol_eo = vol/2;

	myvol = 4*vol_eo;
	setZeroArgs[0] = (void*) &temp;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &temp2;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
	setZeroArgs[0] = (void*) &outVec;
	hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &r; copyArgs[1] = (void*) &inVec;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();
    copyArgs[0] = (void*) &p; copyArgs[1] = (void*) &r;
	hipLaunchCooperativeKernel((void*)&copyVec, dimGrid_zero, dimBlock_copy, copyArgs, 0, NULL);
	hipDeviceSynchronize();


	*dot_res = 0.0;
	myvol = 4*vol_eo;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = abs(*dot_res);


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		myvol = 4*vol_eo;
		setZeroArgs[0] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();
		setZeroArgs[0] = (void*) &temp2;
		hipLaunchCooperativeKernel((void*)&setZeroGPU, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
		hipDeviceSynchronize();

		// Apply D dagger
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Dagger);
		else D.setDagger(MatrixType::Normal);
		D.setInVec(p);
		D.setOutVec(temp2);
		D.applyD();
		// Apply D
		if (Mtype == MatrixType::Normal) D.setDagger(MatrixType::Normal);
		else D.setDagger(MatrixType::Dagger);
		D.setInVec(temp2);
		D.setOutVec(temp);
		D.applyD();
		
		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;

		*dot_res = 0.0;
		myvol = 4*vol_eo;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 


		// x = x + alpha p
		sumArgs[0] = (void*) &outVec;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &outVec;
		sumArgs[3] = (void*) &alpha;
		myvol = vol_eo;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
		
		// r = r - alpha A p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &temp;
		sumArgs[2] = (void*) &r;
		sumArgs[3] = (void*) &alpha;
		alpha = -alpha;
		myvol = vol_eo;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();


		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		*dot_res = 0.0;
		myvol = 4*vol_eo;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;
        rmodsq = abs(*dot_res);

		// p = r - beta p
		sumArgs[0] = (void*) &r;
		sumArgs[1] = (void*) &p;
		sumArgs[2] = (void*) &p;
		sumArgs[3] = (void*) &beta;
		myvol = vol_eo;
		hipLaunchCooperativeKernel((void*)&gpuSumSpinors, dimGrid_dot, dimBlock_dot, sumArgs, 0, NULL);
		hipDeviceSynchronize();
	
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
	if (k >= IterMax) std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";
}*/

__global__ void gpuDotProduct(thrust::complex<double> *vecA, thrust::complex<double> *vecB, thrust::complex<double> *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	extern __shared__ thrust::complex<double> tmp2[];

	thrust::complex<double> temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

	if (tile32.thread_rank() == 0) {
		tmp2[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp2[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}


__global__ void gpuSumSpinors(Spinor<double> *s1, Spinor<double> *s2, Spinor<double> *res, thrust::complex<double> c, int size){
	auto grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < size; i += grid.size()){
		for(int j=0; j<4; j++) res[i].val[j] = s1[i].val[j] + c * s2[i].val[j];
	}
}
