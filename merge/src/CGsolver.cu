#include "hip/hip_runtime.h"
#include "CGsolver.cuh"

CGsolver::CGsolver(int const vol){
    hipMallocManaged(&r, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&p, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&temp2, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&sol, sizeof(Spinor<double>) * vol);
	hipMallocManaged(&dot_res, sizeof(thrust::complex<double>));

    int numBlocks = 0;
	int numThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&numBlocks, &numThreads, gpuDotProduct);
	hipDeviceSynchronize();
	dimGrid_dot = dim3(numBlocks, 1, 1);
	dimBlock_dot = dim3(numThreads, 1, 1);

	numBlocks = 0;
	numThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&numBlocks, &numThreads, setSpinorToZero);
	hipDeviceSynchronize();
	dimGrid_setZero = dim3(numBlocks, 1, 1);
	dimBlock_setZero = dim3(numThreads, 1, 1);

	
}

CGsolver::~CGsolver(){
	hipFree(r);
	hipFree(p);
	hipFree(temp);
	hipFree(temp2);
	hipFree(dot_res);
	hipFree(sol);
}

__global__ void setSpinorToZero(Spinor<double> *s, int const vol){
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();

	thrust::complex<double> im (0.0, 1.0);

	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
		for(int j=0; j<4; j++) s[i].val[j] = 0.0;
	}
}


void CGsolver::solve(Spinor<double>  *inVec, Spinor<double> *outVec, DiracOP<double>& D, thrust::complex<double> *M, int const vol){
	//hipMemcpy(inVec, r, vol * sizeof(Spinor<double>), hipMemcpyHostToDevice);
	thrust::complex<double> alpha; // allocate space ??
	double beta, rmodsq;


	MatrixType dag = MatrixType::Normal;
	int const mySize = 4 * vol;

	void *dotArgs[] = {(void*) &r, (void*) &r, (void*) &dot_res, (void*) &mySize};
	void *setZeroArgs[] = {(void*) temp, (void*) &vol};
	void *diagArgs[] = {(void*)&p, (void*)&temp2, (void*) &D.lattice.vol, (void*) &D.fermion_mass, (void*) &g_coupling, (void*)&dag, (void*)&M};
	void *hoppingArgs[] = {(void*)&p, (void*)&temp2, (void*) &D.lattice.vol, (void*)&dag, (void*)&D.lattice.IUP, (void*)&D.lattice.IDN};

	for(int i=0; i<vol; i++) {
		outVec[i] = Spinor<double> ();
		temp[i] = Spinor<double> ();
		temp2[i] = Spinor<double> ();
		for(int j=0; j<4; j++){
			r[i].val[j] = inVec[i].val[j];
			p[i].val[j] = r[i].val[j];
		}
	}

	setZeroArgs[0] = (void*) &temp;
	hipLaunchCooperativeKernel((void*)&setSpinorToZero, dimGrid_setZero, dimBlock_setZero, setZeroArgs, 0, NULL);
	setZeroArgs[0] = (void*) &temp2;
	hipLaunchCooperativeKernel((void*)&setSpinorToZero, dimGrid_setZero, dimBlock_setZero, setZeroArgs, 0, NULL);
	setZeroArgs[0] = (void*) &sol;
	hipLaunchCooperativeKernel((void*)&setSpinorToZero, dimGrid_setZero, dimBlock_setZero, setZeroArgs, 0, NULL);


	*dot_res = 0.0;
	hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
	hipDeviceSynchronize();
	rmodsq = dot_res->real();


	int k;
	for(k=0; k<IterMax && sqrt(rmodsq) > tolerance; k++){

		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) temp[i].val[j] = 2.0 * p[i].val[j];
		}

		// Set buffers to zero to store the result fo the Dirac operator applied to p
		setZeroArgs[0] = (void*) &temp;
		hipLaunchCooperativeKernel((void*)&setSpinorToZero, dimGrid_setZero, dimBlock_setZero, setZeroArgs, 0, NULL);
		setZeroArgs[0] = (void*) &temp2;
		hipLaunchCooperativeKernel((void*)&setSpinorToZero, dimGrid_setZero, dimBlock_setZero, setZeroArgs, 0, NULL);

		// Apply D dagger
		dag = MatrixType::Dagger;
		diagArgs[0] = (void*) &p; diagArgs[1] = (void*) &temp2;
		hoppingArgs[0] = (void*) &p; hoppingArgs[1] = (void*) &temp2;
		D.applyD(diagArgs, hoppingArgs);
		// Apply D
		dag = MatrixType::Normal;
		diagArgs[0] = (void*) &temp2; diagArgs[1] = (void*) &temp;
		hoppingArgs[0] = (void*) &temp2; hoppingArgs[1] = (void*) &temp;
		D.applyD(diagArgs, hoppingArgs);
		
		dotArgs[0] = (void*) &p; dotArgs[1] = (void*) &temp;

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		alpha = rmodsq / *dot_res; 

		// x = x + alpha p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) outVec[i].val[j] += alpha*p[i].val[j];
		}
		// r = r - alpha A p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) r[i].val[j] -= alpha*temp[i].val[j];
		}

		dotArgs[0] = (void*) &r; dotArgs[1] = (void*) &r;
		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		beta = abs(*dot_res) / rmodsq;

		// p = r - beta p
		for(int i=0; i<vol; i++){
			for(int j=0; j<4; j++) p[i].val[j] = r[i].val[j] + beta*p[i].val[j];
		}

		*dot_res = 0.0;
		hipLaunchCooperativeKernel((void*)&gpuDotProduct, dimGrid_dot, dimBlock_dot, dotArgs, sizeof(thrust::complex<double>) * (32), NULL);
		hipDeviceSynchronize();
		rmodsq = abs(*dot_res);
	}

	//if (k < IterMax) std::cout << "Convergence reached in " << k-1 << " steps \n";
	//else std::cout << "Max. number of iterations reached (" << IterMax << "), final err: " << sqrt(rmodsq) << "\n";

}

__global__ void gpuDotProduct(thrust::complex<double> *vecA, thrust::complex<double> *vecB, thrust::complex<double> *result, int size) {
	cg::thread_block cta = cg::this_thread_block();
	cg::grid_group grid = cg::this_grid();
	//*result = 0.0;
	extern __shared__ thrust::complex<double> tmp[];

	thrust::complex<double> temp_sum = 0.0;
	for (int i = grid.thread_rank(); i < size; i += grid.size()) {
		temp_sum += conj(vecA[i]) * vecB[i];
	}

	cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

	temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

	if (tile32.thread_rank() == 0) {
		tmp[tile32.meta_group_rank()] = temp_sum;
	}

	cg::sync(cta);

	if (tile32.meta_group_rank() == 0) {
		temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
		temp_sum = cg::reduce(tile32, temp_sum, cg::plus<thrust::complex<double>>());

		if (tile32.thread_rank() == 0) {
		atomicAdd(reinterpret_cast<double*>(result), temp_sum.real());
		atomicAdd(reinterpret_cast<double*>(result)+1, temp_sum.imag());
		}
	}
}