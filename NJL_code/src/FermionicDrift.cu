#include "hip/hip_runtime.h"
#include "FermionicDrift.cuh"


extern __constant__ double yukawa_coupling_gpu;
extern __constant__ thrust::complex<double> im_gpu;
extern __constant__ double cutFraction_gpu;

FermionicDrift::FermionicDrift(int const seed) : gen(rd()), dist(0.0, 1.0)
{
	hipMallocManaged(&eobuf, sizeof(cp<double>) * vol);
	hipMallocManaged(&state, sizeof(hiprandState) * spinor_vol);

	int nBlocks = 0;
	int nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, setZero_kernel);
	hipDeviceSynchronize();
  	dimGrid_zero = dim3(nBlocks, 1, 1); 
  	dimBlock_zero = dim3(nThreads, 1, 1); 

	nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, fillNormalRND);
	hipDeviceSynchronize();
  	dimGrid_rnd = dim3(nBlocks, 1, 1); 
  	dimBlock_rnd = dim3(nThreads, 1, 1); 

	nBlocks = 0;
	nThreads = 0;
	hipOccupancyMaxPotentialBlockSize(&nBlocks, &nThreads, random_setup_kernel);
	hipDeviceSynchronize();
  	auto dimGrid_setup = dim3(nBlocks, 1, 1); 
  	auto dimBlock_setup = dim3(nThreads, 1, 1); 
	void* setupArgs[3] = {(void*) &seed, (void*) &state, (void*) &spinor_vol};
	hipLaunchCooperativeKernel((void*)&random_setup_kernel, dimGrid_setup, dimBlock_setup, setupArgs, 0, NULL);
	hipDeviceSynchronize();
	
  	
	setZeroArgs[0] = (void*) &afterCG.data();
    setZeroArgs[1] = (void*) &spinor_vol;

	convArgs[0] = (void*) &eobuf;
    convArgs[1] = (void*) &eobuf;
    
	driftArgs[0] = (void*) &afterCG.data();
    driftArgs[1] = (void*) &noiseVec.data();
    driftArgs[2] = (void*) &noiseVec.data();
    driftArgs[3] = (void*) &N2EO.at; 

	rndArgs[0] = (void*) &noiseVec.data();
	rndArgs[1] = (void*) &state;
	rndArgs[2] = (void*) &spinor_vol;
    
    for(int i=0; i<vol; i++) N2EO.at[i] = convertNormalToEO(i);

}

__global__ void random_setup_kernel(int const seed, hiprandState *state, int const vol) {
	cg::grid_group grid = cg::this_grid();
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
		hiprand_init(seed, i, 0, &state[i]);
	}
}

void FermionicDrift::getForce(double *outVec, DiracOP<double>& D, CGsolver& CG, dim3 dimGrid_drift, dim3 dimBlock_drift){
	
	hipLaunchCooperativeKernel((void*)&fillNormalRND, dimGrid_rnd, dimBlock_rnd, rndArgs, 0, NULL);
	hipDeviceSynchronize();
	
	// set some spinors to zero
	setZeroArgs[0] = (void*)&afterCG.data();
	hipLaunchCooperativeKernel((void*)&setZero_kernel, dimGrid_zero, dimBlock_zero, setZeroArgs, 0, NULL);
	hipDeviceSynchronize();

	switch (CGmode){
		case '0':
			CG.solve(noiseVec.data(), buf.data(), D, MatrixType::Dagger);
			D.applyD(buf.data(), afterCG.data(), MatrixType::Normal);
			hipDeviceSynchronize();
			break;
		case '1':
			
			//Dirac_d.D_oo_inv(psiField.pos.begin() + lattice.vol/2, temp2.begin());
			//Dirac_d.D_eo(temp2.begin(), temp1.begin());
			/*D.setInVec(); D.setOutVec();
			hipLaunchCooperativeKernel((void*)&D_oo_inv<T>, dimGrid_Doo_inv, dimBlock_Doo_inv, diagArgs, 0, NULL);
    		hipDeviceSynchronize();
			hipLaunchCooperativeKernel((void*)&D_eo<T>, dimGrid_Deo, dimBlock_Deo, diagArgs, 0, NULL);
    		hipDeviceSynchronize();

			spinorDiff(psiField.pos.begin(), psiField.pos.begin() + lattice.vol/2, temp1.begin(), temp2.begin());
			
			CGdouble.solve_Dhat(temp2.begin(), temp2.end(), temp3.begin());
			for(int i=0; i<lattice.vol/2; i++) std::fill(psiField.pos[i].val.begin(), psiField.pos[i].val.begin(), 0.0);
			Dirac_d.applyDhatTo(temp3.begin(), psiField.pos.begin(), MatrixType::Dagger);

			std::fill(temp1.begin(), temp1.end(), Spinor_d());
			Dirac_d.D_oe(psiField.pos.begin(), temp1.begin());

			spinorDiff(psiField.pos.begin() + lattice.vol/2, psiField.pos.end(), temp1.begin(), temp3.begin());

			for(int i=lattice.vol/2; i<lattice.vol; i++) std::fill(psiField.pos[i].val.begin(), psiField.pos[i].val.end(), 0.0);
			Dirac_d.D_oo_inv(temp3.begin(), psiField.pos.begin() + lattice.vol/2);*/
			break;
		}
		
	driftArgs[0] = (void*) &afterCG.data();
	driftArgs[1] = (void*) &noiseVec.data();
	driftArgs[2] = (void*) &outVec;
	hipLaunchCooperativeKernel((void*)&computeDrift, dimGrid_drift, dimBlock_drift, driftArgs, 0, NULL);
	hipDeviceSynchronize();
	 
}


__global__ void computeDrift(cp<double> *afterCG,cp<double> *noise, double *outVec, int *N2EO){

	cg::grid_group grid = cg::this_grid();
	int eo_i;
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){
        eo_i = N2EO[i];
		outVec[i] = - yukawa_coupling_gpu * ( conj(afterCG[4*eo_i+0])*noise[4*eo_i+0]
                                            + conj(afterCG[4*eo_i+1])*noise[4*eo_i+1] 
                                            + conj(afterCG[4*eo_i+2])*noise[4*eo_i+2] 
                                            + conj(afterCG[4*eo_i+3])*noise[4*eo_i+3]).real();
	}

}

__global__ void fillNormalRND(cp<double>* vec, hiprandState *state, int const vol){
	cg::grid_group grid = cg::this_grid();
	auto myState = state[grid.thread_rank()];
	for (int i = grid.thread_rank(); i < vol; i += grid.size()){	 
		myState = state[i]; 
		vec[i] = hiprand_normal_double(&myState); 
		state[i] = myState;
	}
}

